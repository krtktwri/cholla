
#include <hip/hip_runtime.h>
/*! \file hydro_cuda.cu
 *  \brief Definitions of functions used in all cuda integration algorithms. */
#ifdef CUDA

#include <stdio.h>
#include <math.h>
#include <float.h>

#include "../utils/gpu.hpp"
#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../hydro/hydro_cuda.h"
#include "../gravity/gravity_cuda.h"
#include "../utils/hydro_utilities.h"
#include "../utils/cuda_utilities.h"
#include "../utils/reduction_utilities.h"


__global__ void Update_Conserved_Variables_1D(Real *dev_conserved, Real *dev_F, int n_cells, int x_off, int n_ghost, Real dx, Real xbound, Real dt, Real gamma, int n_fields)
{
  int id;
  #ifdef STATIC_GRAV
  Real d, d_inv, vx;
  Real gx, d_n, d_inv_n, vx_n;
  gx = 0.0;
  #endif

  Real dtodx = dt/dx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;


  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    #ifdef STATIC_GRAV
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    #endif

    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F[            id-1] - dev_F[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F[  n_cells + id-1] - dev_F[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F[2*n_cells + id-1] - dev_F[2*n_cells + id]);
    dev_conserved[3*n_cells + id] += dtodx * (dev_F[3*n_cells + id-1] - dev_F[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F[4*n_cells + id-1] - dev_F[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved[(5+i)*n_cells + id] += dtodx * (dev_F[(5+i)*n_cells + id-1] - dev_F[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] += dtodx * (dev_F[(n_fields-1)*n_cells + id-1] - dev_F[(n_fields-1)*n_cells + id]);
    #endif
    #ifdef STATIC_GRAV // add gravitational source terms, time averaged from n to n+1
    calc_g_1D(id, x_off, n_ghost, dx, xbound, &gx);
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[4*n_cells + id] += 0.25*dt*gx*(d + d_n)*(vx + vx_n);
    #endif
    if (dev_conserved[id] != dev_conserved[id]) printf("%3d Thread crashed in final update. %f\n", id, dev_conserved[id]);
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0) printf("%d Negative pressure after final update.\n", id);
    */
  }


}


__global__ void Update_Conserved_Variables_2D(Real *dev_conserved, Real *dev_F_x, Real *dev_F_y, int nx, int ny, int x_off, int y_off, int n_ghost, Real dx, Real dy, Real xbound, Real ybound, Real dt, Real gamma, int n_fields)
{
  int id, xid, yid, n_cells;
  int imo, jmo;

  #ifdef STATIC_GRAV
  Real d, d_inv, vx, vy;
  Real gx, gy, d_n, d_inv_n, vx_n, vy_n;
  gx = 0.0;
  gy = 0.0;
  #endif

  Real dtodx = dt/dx;
  Real dtody = dt/dy;

  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  imo = xid-1 + yid*nx;
  jmo = xid + (yid-1)*nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    #ifdef STATIC_GRAV
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    #endif
    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                  +  dtody * (dev_F_y[            jmo] - dev_F_y[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                                  +  dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                                  +  dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]);
    dev_conserved[3*n_cells + id] += dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                  +  dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                  +  dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved[(5+i)*n_cells + id] += dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                        +  dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] += dtodx * (dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                                  +  dtody * (dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id]);
    #endif
    #ifdef STATIC_GRAV
    // calculate the gravitational acceleration as a function of x & y position
    calc_g_2D(xid, yid, x_off, y_off, n_ghost, dx, dy, xbound, ybound, &gx, &gy);
    // add gravitational source terms, time averaged from n to n+1
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    vy_n =  dev_conserved[2*n_cells + id] * d_inv_n;
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[2*n_cells + id] += 0.5*dt*gy*(d + d_n);
    dev_conserved[4*n_cells + id] += 0.25*dt*gx*(d + d_n)*(vx + vx_n)
                                  +  0.25*dt*gy*(d + d_n)*(vy + vy_n);
    #endif
    if (dev_conserved[id] < 0.0 || dev_conserved[id] != dev_conserved[id]) {
      printf("%3d %3d Thread crashed in final update. %f %f %f\n", xid, yid, dtodx*(dev_F_x[imo]-dev_F_x[id]), dtody*(dev_F_y[jmo]-dev_F_y[id]), dev_conserved[id]);
    }
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0)
      printf("%3d %3d Negative pressure after final update. %f %f %f %f\n", xid, yid, dev_conserved[4*n_cells + id], 0.5*d*vx*vx, 0.5*d*vy*vy, P);
    */
  }

}



__global__ void Update_Conserved_Variables_3D(Real *dev_conserved,
                                              Real *Q_Lx, Real *Q_Rx, Real *Q_Ly, Real *Q_Ry, Real *Q_Lz, Real *Q_Rz,
                                              Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z,
                                              int nx, int ny, int nz, int x_off, int y_off, int z_off, int n_ghost,
                                              Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real dt,
                                              Real gamma, int n_fields, Real density_floor, Real *dev_potential )
{
  int id, xid, yid, zid, n_cells;
  int imo, jmo, kmo;

  #ifdef STATIC_GRAV
  Real d, d_inv, vx, vy, vz;
  Real gx, gy, gz, d_n, d_inv_n, vx_n, vy_n, vz_n;
  gx = 0.0;
  gy = 0.0;
  gz = 0.0;
  #endif

  #ifdef DENSITY_FLOOR
  Real dens_0;
  #endif

  #ifdef GRAVITY
  Real d, d_inv, vx, vy, vz;
  Real gx, gy, gz, d_n, d_inv_n, vx_n, vy_n, vz_n;
  Real pot_l, pot_r;
  int id_l, id_r;
  gx = 0.0;
  gy = 0.0;
  gz = 0.0;

  #ifdef GRAVITY_5_POINTS_GRADIENT
  int id_ll, id_rr;
  Real pot_ll, pot_rr;
  #endif

  #endif //GRAVITY

  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;
  imo = xid-1 + yid*nx + zid*nx*ny;
  jmo = xid + (yid-1)*nx + zid*nx*ny;
  kmo = xid + yid*nx + (zid-1)*nx*ny;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    #if defined(STATIC_GRAV) ||  defined(GRAVITY)
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    #endif

    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                  +  dtody * (dev_F_y[            jmo] - dev_F_y[            id])
                                  +  dtodz * (dev_F_z[            kmo] - dev_F_z[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                                  +  dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id])
                                  +  dtodz * (dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                                  +  dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id])
                                  +  dtodz * (dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id]);
    dev_conserved[3*n_cells + id] += dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                  +  dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
                                  +  dtodz * (dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                  +  dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id])
                                  +  dtodz * (dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved[(5+i)*n_cells + id] += dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                    +  dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id])
                                    +  dtodz * (dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id]);
      #ifdef COOLING_GRACKLE
      // If the updated value is negative, then revert to the value before the update
      if ( dev_conserved[(5+i)*n_cells + id] < 0 ){
        dev_conserved[(5+i)*n_cells + id] -= dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                      +  dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id])
                                      +  dtodz * (dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id]);
      }
      #endif
    }
    #endif
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] += dtodx * (dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                                  +  dtody * (dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id])
                                  +  dtodz * (dev_F_z[(n_fields-1)*n_cells + kmo] - dev_F_z[(n_fields-1)*n_cells + id]);
                                  // +  0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo) + dtodz*(vz_kmo-vz_kpo));
                                  //Note: this term is added in a separate kernel to avoid synchronization issues
    #endif

    #ifdef DENSITY_FLOOR
    if ( dev_conserved[            id] < density_floor ){
      if (dev_conserved[            id] > 0){
        dens_0 = dev_conserved[            id];
        // Set the density to the density floor
        dev_conserved[            id] = density_floor;
        // Scale the conserved values to the new density
        dev_conserved[1*n_cells + id] *= (density_floor / dens_0);
        dev_conserved[2*n_cells + id] *= (density_floor / dens_0);
        dev_conserved[3*n_cells + id] *= (density_floor / dens_0);
        dev_conserved[4*n_cells + id] *= (density_floor / dens_0);
        #ifdef DE
        dev_conserved[(n_fields-1)*n_cells + id] *= (density_floor / dens_0);
        #endif
      }
      else{
        // If the density is negative: average the density on that cell
        dens_0 = dev_conserved[            id];
        Average_Cell_Single_Field( 0, xid, yid, zid, nx, ny, nz, n_cells, dev_conserved );
      }
    }
    #endif//DENSITY_FLOOR

    #ifdef STATIC_GRAV
    calc_g_3D(xid, yid, zid, x_off, y_off, z_off, n_ghost, dx, dy, dz, xbound, ybound, zbound, &gx, &gy, &gz);
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    vy_n =  dev_conserved[2*n_cells + id] * d_inv_n;
    vz_n =  dev_conserved[3*n_cells + id] * d_inv_n;
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[2*n_cells + id] += 0.5*dt*gy*(d + d_n);
    dev_conserved[3*n_cells + id] += 0.5*dt*gz*(d + d_n);
    dev_conserved[4*n_cells + id] += 0.25*dt*gx*(d + d_n)*(vx + vx_n)
                                  +  0.25*dt*gy*(d + d_n)*(vy + vy_n)
                                  +  0.25*dt*gz*(d + d_n)*(vz + vz_n);
    #endif

    #ifdef GRAVITY
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    vy_n =  dev_conserved[2*n_cells + id] * d_inv_n;
    vz_n =  dev_conserved[3*n_cells + id] * d_inv_n;

    // Calculate the -gradient of potential
    // Get X componet of gravity field
    id_l = (xid-1) + (yid)*nx + (zid)*nx*ny;
    id_r = (xid+1) + (yid)*nx + (zid)*nx*ny;
    pot_l = dev_potential[id_l];
    pot_r = dev_potential[id_r];
    #ifdef GRAVITY_5_POINTS_GRADIENT
    id_ll = (xid-2) + (yid)*nx + (zid)*nx*ny;
    id_rr = (xid+2) + (yid)*nx + (zid)*nx*ny;
    pot_ll = dev_potential[id_ll];
    pot_rr = dev_potential[id_rr];
    gx = -1 * ( -pot_rr + 8*pot_r - 8*pot_l + pot_ll) / (12*dx);
    #else
    gx = -0.5*( pot_r - pot_l ) / dx;
    #endif

    //Get Y componet of gravity field
    id_l = (xid) + (yid-1)*nx + (zid)*nx*ny;
    id_r = (xid) + (yid+1)*nx + (zid)*nx*ny;
    pot_l = dev_potential[id_l];
    pot_r = dev_potential[id_r];
    #ifdef GRAVITY_5_POINTS_GRADIENT
    id_ll = (xid) + (yid-2)*nx + (zid)*nx*ny;
    id_rr = (xid) + (yid+2)*nx + (zid)*nx*ny;
    pot_ll = dev_potential[id_ll];
    pot_rr = dev_potential[id_rr];
    gy = -1 * ( -pot_rr + 8*pot_r - 8*pot_l + pot_ll) / (12*dx);
    #else
    gy = -0.5*( pot_r - pot_l ) / dy;
    #endif
    //Get Z componet of gravity field
    id_l = (xid) + (yid)*nx + (zid-1)*nx*ny;
    id_r = (xid) + (yid)*nx + (zid+1)*nx*ny;
    pot_l = dev_potential[id_l];
    pot_r = dev_potential[id_r];
    #ifdef GRAVITY_5_POINTS_GRADIENT
    id_ll = (xid) + (yid)*nx + (zid-2)*nx*ny;
    id_rr = (xid) + (yid)*nx + (zid+2)*nx*ny;
    pot_ll = dev_potential[id_ll];
    pot_rr = dev_potential[id_rr];
    gz = -1 * ( -pot_rr + 8*pot_r - 8*pot_l + pot_ll) / (12*dx);
    #else
    gz = -0.5*( pot_r - pot_l ) / dz;
    #endif

    //Add gravity term to Momentum
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[2*n_cells + id] += 0.5*dt*gy*(d + d_n);
    dev_conserved[3*n_cells + id] += 0.5*dt*gz*(d + d_n);

    //Add gravity term to Total Energy
    //Add the work done by the gravitational force
    dev_conserved[4*n_cells + id] += 0.5* dt * ( gx*(d*vx + d_n*vx_n) +  gy*(d*vy + d_n*vy_n) +  gz*(d*vz + d_n*vz_n) );

    #endif


    #if !( defined(DENSITY_FLOOR) && defined(TEMPERATURE_FLOOR) )
    if (dev_conserved[id] < 0.0 || dev_conserved[id] != dev_conserved[id] || dev_conserved[4*n_cells + id] < 0.0 || dev_conserved[4*n_cells+id] != dev_conserved[4*n_cells+id]) {
      printf("%3d %3d %3d Thread crashed in final update. %e %e %e %e %e\n", xid+x_off, yid+y_off, zid+z_off, dev_conserved[id], dtodx*(dev_F_x[imo]-dev_F_x[id]), dtody*(dev_F_y[jmo]-dev_F_y[id]), dtodz*(dev_F_z[kmo]-dev_F_z[id]), dev_conserved[4*n_cells+id]);
    }
    #endif//DENSITY_FLOOR
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0) printf("%3d %3d %3d Negative pressure after final update. %f %f %f %f %f\n", xid, yid, zid, dev_conserved[4*n_cells + id], 0.5*d*vx*vx, 0.5*d*vy*vy, 0.5*d*vz*vz, P);
    */
  }

}

 __device__ __host__ Real hydroInverseCrossingTime(Real const &E,
                                                   Real const &d,
                                                   Real const &d_inv,
                                                   Real const &vx,
                                                   Real const &vy,
                                                   Real const &vz,
                                                   Real const &dx,
                                                   Real const &dy,
                                                   Real const &dz,
                                                   Real const &gamma)
{
  // Compute pressure and sound speed
  Real P  = (E - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
  Real cs = sqrt(d_inv * gamma * P);

  // Find maximum inverse crossing time in the cell (i.e. minimum crossing time)
  Real cellMaxInverseDt = fmax((fabs(vx)+cs)/dx, (fabs(vy)+cs)/dy);
  cellMaxInverseDt      = fmax(cellMaxInverseDt, (fabs(vz)+cs)/dz);
  cellMaxInverseDt      = fmax(cellMaxInverseDt, 0.0);

  return cellMaxInverseDt;
}

__device__ __host__ Real mhdInverseCrossingTime(Real const &E,
                                                Real const &d,
                                                Real const &d_inv,
                                                Real const &vx,
                                                Real const &vy,
                                                Real const &vz,
                                                Real const &avgBx,
                                                Real const &avgBy,
                                                Real const &avgBz,
                                                Real const &dx,
                                                Real const &dy,
                                                Real const &dz,
                                                Real const &gamma)
{
  // Compute the gas pressure and fast magnetosonic speed
  Real gasP = mhdUtils::computeGasPressure(E, d, vx*d, vy*d, vz*d, avgBx, avgBy, avgBz, gamma);
  Real cf   = mhdUtils::fastMagnetosonicSpeed(d, gasP, avgBx, avgBy, avgBz, gamma);

  // Find maximum inverse crossing time in the cell (i.e. minimum crossing time)
  Real cellMaxInverseDt = fmax((fabs(vx)+cf)/dx, (fabs(vy)+cf)/dy);
  cellMaxInverseDt      = fmax(cellMaxInverseDt, (fabs(vz)+cf)/dz);
  cellMaxInverseDt      = fmax(cellMaxInverseDt, 0.0);

  return cellMaxInverseDt;
}



__global__ void Calc_dt_1D(Real *dev_conserved, Real *dev_dti, Real gamma, int n_ghost, int nx, Real dx)
{
  Real max_dti = -DBL_MAX;

  Real d, d_inv, vx, vy, vz, P, cs;
  int n_cells = nx;

  // Grid stride loop to perform as much of the reduction as possible. The
  // fact that `id` has type `size_t` is important. I'm not totally sure why
  // but setting it to int results in some kind of silent over/underflow issue
  // even though we're not hitting those kinds of numbers. Setting it to type
  // uint or size_t fixes them
  for(size_t id = threadIdx.x + blockIdx.x * blockDim.x; id < n_cells; id += blockDim.x * gridDim.x)
  {
    // threads corresponding to real cells do the calculation
    if (id > n_ghost - 1 && id < n_cells-n_ghost)
    {
      // start timestep calculation here
      // every thread collects the conserved variables it needs from global memory
      d  =  dev_conserved[            id];
      d_inv = 1.0 / d;
      vx =  dev_conserved[1*n_cells + id] * d_inv;
      vy =  dev_conserved[2*n_cells + id] * d_inv;
      vz =  dev_conserved[3*n_cells + id] * d_inv;
      P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
      P  = fmax(P, (Real) TINY_NUMBER);
      // find the max wavespeed in that cell, use it to calculate the inverse timestep
      cs = sqrt(d_inv * gamma * P);
      max_dti = fmax(max_dti,(fabs(vx)+cs)/dx);
    }
  }

  // do the block wide reduction (find the max inverse timestep in the block)
  // then write it to that block's location in the dev_dti array
  max_dti = reduction_utilities::blockReduceMax(max_dti);
  if (threadIdx.x == 0) dev_dti[blockIdx.x] = max_dti;
}



__global__ void Calc_dt_2D(Real *dev_conserved, Real *dev_dti, Real gamma, int n_ghost, int nx, int ny, Real dx, Real dy)
{
  Real max_dti = -DBL_MAX;

  Real d, d_inv, vx, vy, vz, P, cs;
  int xid, yid, n_cells;
  n_cells = nx*ny;

  // Grid stride loop to perform as much of the reduction as possible. The
  // fact that `id` has type `size_t` is important. I'm not totally sure why
  // but setting it to int results in some kind of silent over/underflow issue
  // even though we're not hitting those kinds of numbers. Setting it to type
  // uint or size_t fixes them
  for(size_t id = threadIdx.x + blockIdx.x * blockDim.x; id < n_cells; id += blockDim.x * gridDim.x)
  {
    // get a global thread ID
    yid = id / nx;
    xid = id - yid*nx;

    // threads corresponding to real cells do the calculation
    if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
    {
      // every thread collects the conserved variables it needs from global memory
      d  =  dev_conserved[            id];
      d_inv = 1.0 / d;
      vx =  dev_conserved[1*n_cells + id] * d_inv;
      vy =  dev_conserved[2*n_cells + id] * d_inv;
      vz =  dev_conserved[3*n_cells + id] * d_inv;
      P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
      P  = fmax(P, (Real) 1.0e-20);
      // find the max wavespeed in that cell, use it to calculate the inverse timestep
      cs = sqrt(d_inv * gamma * P);
      max_dti = fmax(max_dti,fmax((fabs(vx)+cs)/dx, (fabs(vy)+cs)/dy));
    }
  }

  // do the block wide reduction (find the max inverse timestep in the block)
  // then write it to that block's location in the dev_dti array
  max_dti = reduction_utilities::blockReduceMax(max_dti);
  if (threadIdx.x == 0) dev_dti[blockIdx.x] = max_dti;
}


__global__ void Calc_dt_3D(Real *dev_conserved, Real *dev_dti, Real gamma, int n_ghost, int n_fields, int nx, int ny, int nz, Real dx, Real dy, Real dz)
{
  Real max_dti = -DBL_MAX;

  Real d, d_inv, vx, vy, vz, E;
  #ifdef  MHD
    Real avgBx, avgBy, avgBz;
  #endif  //MHD
  int xid, yid, zid, n_cells;

  n_cells = nx*ny*nz;

  // Grid stride loop to perform as much of the reduction as possible. The
  // fact that `id` has type `size_t` is important. I'm not totally sure why
  // but setting it to int results in some kind of silent over/underflow issue
  // even though we're not hitting those kinds of numbers. Setting it to type
  // uint or size_t fixes them
  for(size_t id = threadIdx.x + blockIdx.x * blockDim.x; id < n_cells; id += blockDim.x * gridDim.x)
  {
    // get a global thread ID
    cuda_utilities::compute3DIndices(id, nx, ny, xid, yid, zid);

    // threads corresponding to real cells do the calculation
    if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
    {
      // every thread collects the conserved variables it needs from global memory
      d     = dev_conserved[            id];
      d_inv = 1.0 / d;
      vx    = dev_conserved[1*n_cells + id] * d_inv;
      vy    = dev_conserved[2*n_cells + id] * d_inv;
      vz    = dev_conserved[3*n_cells + id] * d_inv;
      E     = dev_conserved[4*n_cells + id];
      #ifdef  MHD
        // Compute the cell centered magnetic field using a straight average of
        // the faces
        mhdUtils::cellCenteredMagneticFields(dev_conserved, id, xid, yid, zid, n_cells, nx, ny, avgBx, avgBy, avgBz);
      #endif  //MHD

      // Compute the maximum inverse crossing time in the cell
      #ifdef  MHD
        max_dti = fmax(max_dti,mhdInverseCrossingTime(E, d, d_inv, vx, vy, vz, avgBx, avgBy, avgBz, dx, dy, dz, gamma));
      #else  // not MHD
        max_dti = fmax(max_dti,hydroInverseCrossingTime(E, d, d_inv, vx, vy, vz, dx, dy, dz, gamma));
      #endif  //MHD

    }
  }

  // do the block wide reduction (find the max inverse timestep in the block)
  // then write it to that block's location in the dev_dti array
  max_dti = reduction_utilities::blockReduceMax(max_dti);
  if (threadIdx.x == 0) dev_dti[blockIdx.x] = max_dti;
}

Real Calc_dt_GPU(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dx, Real dy, Real dz, Real gamma )
{
  // set values for GPU kernels
  uint threadsPerBlock, numBlocks;
  int ngrid = (nx*ny*nz + TPB - 1 )/TPB;
  // reduction_utilities::reductionLaunchParams(numBlocks, threadsPerBlock); // Uncomment this if we fix the AtomicDouble bug - Alwin
  threadsPerBlock = TPB;
  numBlocks = ngrid;

  Real* dev_dti = dev_dti_array;


  // compute dt and store in dev_dti
  if (nx > 1 && ny == 1 && nz == 1) //1D
  {
    hipLaunchKernelGGL(Calc_dt_1D, numBlocks, threadsPerBlock, 0, 0, dev_conserved, dev_dti, gamma, n_ghost, nx, dx);
  }
  else if (nx > 1 && ny > 1 && nz == 1) //2D
  {
    hipLaunchKernelGGL(Calc_dt_2D, numBlocks, threadsPerBlock, 0, 0, dev_conserved, dev_dti, gamma, n_ghost, nx, ny, dx, dy);
  }
  else if (nx > 1 && ny > 1 && nz > 1) //3D
  {
    hipLaunchKernelGGL(Calc_dt_3D, numBlocks, threadsPerBlock, 0, 0, dev_conserved, dev_dti, gamma, n_ghost, n_fields, nx, ny, nz, dx, dy, dz);
  }
  CudaCheckError();

  Real max_dti=0;

  /* Uncomment the below if we fix the AtomicDouble bug - Alwin
  // copy device side max_dti to host side max_dti


  CudaSafeCall( cudaMemcpy(&max_dti, dev_dti, sizeof(Real), cudaMemcpyDeviceToHost) );
  cudaDeviceSynchronize();

  return max_dti;
  */

  int dev_dti_length = numBlocks;
  CudaSafeCall(cudaMemcpy(host_dti_array,dev_dti, dev_dti_length*sizeof(Real), cudaMemcpyDeviceToHost));
  cudaDeviceSynchronize();

  for (int i=0;i<dev_dti_length;i++){
    max_dti = fmax(max_dti,host_dti_array[i]);
  }

  return max_dti;
}


#ifdef AVERAGE_SLOW_CELLS

void Average_Slow_Cells( Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dx, Real dy, Real dz, Real gamma, Real max_dti_slow ){

  // set values for GPU kernels
  int n_cells = nx*ny*nz;
  int ngrid = (n_cells + TPB - 1) / TPB;
  // number of blocks per 1D grid
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block
  dim3 dim1dBlock(TPB, 1, 1);

  if (nx > 1 && ny > 1 && nz > 1){ //3D
    hipLaunchKernelGGL(Average_Slow_Cells_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost, n_fields, dx, dy, dz, gamma, max_dti_slow );
  }
}

__global__ void Average_Slow_Cells_3D(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dx, Real dy, Real dz, Real gamma, Real max_dti_slow ){

  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, E, max_dti;
  #ifdef  MHD
    Real avgBx, avgBy, avgBz;
  #endif  //MHD

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  n_cells = nx*ny*nz;

  cuda_utilities::compute3DIndices(id, nx, ny, xid, yid, zid);


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];

    #ifdef  MHD
      // Compute the cell centered magnetic field using a straight average of the faces
      mhdUtils::cellCenteredMagneticFields(dev_conserved, id, xid, yid, zid, n_cells, nx, ny, avgBx, avgBy, avgBz);
    #endif  //MHD

    // Compute the maximum inverse crossing time in the cell
    #ifdef  MHD
      max_dti = mhdInverseCrossingTime(E, d, d_inv, vx, vy, vz, avgBx, avgBy, avgBz, dx, dy, dz, gamma);
    #else  // not MHD
      max_dti = hydroInverseCrossingTime(E, d, d_inv, vx, vy, vz, dx, dy, dz, gamma);
    #endif  //MHD

    if (max_dti > max_dti_slow){
      // Average this cell
      printf(" Average Slow Cell [ %d %d %d ] -> dt_cell=%f    dt_min=%f\n", xid, yid, zid, 1./max_dti,  1./max_dti_slow );
      Average_Cell_All_Fields( xid, yid, zid, nx, ny, nz, n_cells, n_fields, dev_conserved );
    }
  }
}
#endif //AVERAGE_SLOW_CELLS


#ifdef DE
__global__ void Partial_Update_Advected_Internal_Energy_1D( Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, int nx, int n_ghost, Real dx, Real dt, Real gamma, int n_fields ){

  int id, xid, n_cells;
  int imo, ipo;
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo;
  Real  P, E, E_kin, GE;


  Real dtodx = dt/dx;
  n_cells = nx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  xid = id;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    GE = dev_conserved[(n_fields-1)*n_cells + id];
    E_kin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    P = hydro_utilities::Get_Pressure_From_DE( E, E - E_kin, GE, gamma );
    P  = fmax(P, (Real) TINY_NUMBER);

    imo = xid-1;
    ipo = xid+1;

    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo];
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo];

    // Use center values of neighbor cells for the divergence of velocity
    dev_conserved[(n_fields-1)*n_cells + id] += 0.5*P*(dtodx*(vx_imo-vx_ipo));

  }
}


__global__ void Partial_Update_Advected_Internal_Energy_2D( Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, Real *Q_Ly, Real *Q_Ry, int nx, int ny, int n_ghost, Real dx, Real dy, Real dt, Real gamma, int n_fields ){

  int id, xid, yid, n_cells;
  int imo, jmo;
  int ipo, jpo;
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo;
  Real  P, E, E_kin, GE;


  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    GE = dev_conserved[(n_fields-1)*n_cells + id];
    E_kin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    P = hydro_utilities::Get_Pressure_From_DE( E, E - E_kin, GE, gamma );
    P  = fmax(P, (Real) TINY_NUMBER);

    imo = xid-1 + yid*nx;
    ipo = xid+1 + yid*nx;
    jmo = xid + (yid-1)*nx;
    jpo = xid + (yid+1)*nx;

    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo];
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo];
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo];
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo];

    // Use center values of neighbor cells for the divergence of velocity
    dev_conserved[(n_fields-1)*n_cells + id] += 0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo));

  }
}

__global__ void Partial_Update_Advected_Internal_Energy_3D( Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, Real *Q_Ly, Real *Q_Ry, Real *Q_Lz, Real *Q_Rz, int nx, int ny, int nz,  int n_ghost, Real dx, Real dy, Real dz,  Real dt, Real gamma, int n_fields ){

  int id, xid, yid, zid, n_cells;
  int imo, jmo, kmo;
  int ipo, jpo, kpo;
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo, vz_kmo, vz_kpo;
  Real  P, E, E_kin, GE;
  // Real vx_L, vx_R, vy_L, vy_R, vz_L, vz_R;


  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    GE = dev_conserved[(n_fields-1)*n_cells + id];
    E_kin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    P = hydro_utilities::Get_Pressure_From_DE( E, E - E_kin, GE, gamma );
    P  = fmax(P, (Real) TINY_NUMBER);

    imo = xid-1 + yid*nx + zid*nx*ny;
    jmo = xid + (yid-1)*nx + zid*nx*ny;
    kmo = xid + yid*nx + (zid-1)*nx*ny;

    ipo = xid+1 + yid*nx + zid*nx*ny;
    jpo = xid + (yid+1)*nx + zid*nx*ny;
    kpo = xid + yid*nx + (zid+1)*nx*ny;

    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo];
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo];
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo];
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo];
    vz_kmo = dev_conserved[3*n_cells + kmo] / dev_conserved[kmo];
    vz_kpo = dev_conserved[3*n_cells + kpo] / dev_conserved[kpo];

    // Use center values of neighbor cells for the divergence of velocity
    dev_conserved[(n_fields-1)*n_cells + id] += 0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo) + dtodz*(vz_kmo-vz_kpo));

    // OPTION 2: Use the reconstructed velocities to compute the velocity gradient
    //Use the reconstructed Velocities instead of neighbor cells centered values
    // vx_R = Q_Lx[1*n_cells + id]  / Q_Lx[id];
    // vx_L = Q_Rx[1*n_cells + imo] / Q_Rx[imo];
    // vy_R = Q_Ly[2*n_cells + id]  / Q_Ly[id];
    // vy_L = Q_Ry[2*n_cells + jmo] / Q_Ry[jmo];
    // vz_R = Q_Lz[3*n_cells + id]  / Q_Lz[id];
    // vz_L = Q_Rz[3*n_cells + kmo] / Q_Rz[kmo];

    //Use the reconstructed Velocities instead of neighbor cells centered values
    // dev_conserved[(n_fields-1)*n_cells + id] +=  P * ( dtodx * ( vx_L - vx_R ) + dtody * ( vy_L - vy_R ) + dtodz * ( vz_L - vz_R ) );


  }
}


__global__ void Select_Internal_Energy_1D( Real *dev_conserved, int nx, int n_ghost, int n_fields ){

  int id, xid, n_cells;
  Real d, d_inv, vx, vy, vz, E, U_total, U_advected, U, Emax;
  int imo, ipo;
  n_cells = nx;

  Real eta_2 = DE_ETA_2;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  xid = id;

  imo = max(xid-1, n_ghost);
  ipo = min(xid+1, nx-n_ghost-1);


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    U_advected = dev_conserved[(n_fields-1)*n_cells + id];
    U_total = E - 0.5*d*( vx*vx + vy*vy + vz*vz );

    //find the max nearby total energy
    Emax = fmax(dev_conserved[4*n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4*n_cells + ipo]);

    if (U_total/Emax > eta_2 ) U = U_total;
    else U = U_advected;

    //Optional: Avoid Negative Internal  Energies
    U = fmax(U, (Real) TINY_NUMBER);

    //Write Selected internal energy to the GasEnergy array ONLY
    //to avoid mixing updated and non-updated values of E
    //since the Dual Energy condition depends on the neighbor cells
    dev_conserved[(n_fields-1)*n_cells + id] = U;

  }
}


__global__ void Select_Internal_Energy_2D( Real *dev_conserved, int nx, int ny, int n_ghost, int n_fields ){

  int id, xid, yid, n_cells;
  Real d, d_inv, vx, vy, vz, E, U_total, U_advected, U, Emax;
  int imo, ipo, jmo, jpo;
  n_cells = nx*ny;

  Real eta_2 = DE_ETA_2;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;

  imo = max(xid-1, n_ghost);
  imo = imo + yid*nx;
  ipo = min(xid+1, nx-n_ghost-1);
  ipo = ipo + yid*nx;
  jmo = max(yid-1, n_ghost);
  jmo = xid + jmo*nx;
  jpo = min(yid+1, ny-n_ghost-1);
  jpo = xid + jpo*nx;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    U_advected = dev_conserved[(n_fields-1)*n_cells + id];
    U_total = E - 0.5*d*( vx*vx + vy*vy + vz*vz );

    //find the max nearby total energy
    Emax = fmax(dev_conserved[4*n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4*n_cells + ipo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jmo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jpo]);

    if (U_total/Emax > eta_2 ) U = U_total;
    else U = U_advected;

    //Optional: Avoid Negative Internal  Energies
    U = fmax(U, (Real) TINY_NUMBER);

    //Write Selected internal energy to the GasEnergy array ONLY
    //to avoid mixing updated and non-updated values of E
    //since the Dual Energy condition depends on the neighbour cells
    dev_conserved[(n_fields-1)*n_cells + id] = U;

  }
}


__global__ void Select_Internal_Energy_3D( Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields ){

  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, E, U_total, U_advected, U, Emax;
  int imo, ipo, jmo, jpo, kmo, kpo;
  n_cells = nx*ny*nz;

  Real eta_2 = DE_ETA_2;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;

  imo = max(xid-1, n_ghost);
  imo = imo + yid*nx + zid*nx*ny;
  ipo = min(xid+1, nx-n_ghost-1);
  ipo = ipo + yid*nx + zid*nx*ny;
  jmo = max(yid-1, n_ghost);
  jmo = xid + jmo*nx + zid*nx*ny;
  jpo = min(yid+1, ny-n_ghost-1);
  jpo = xid + jpo*nx + zid*nx*ny;
  kmo = max(zid-1, n_ghost);
  kmo = xid + yid*nx + kmo*nx*ny;
  kpo = min(zid+1, nz-n_ghost-1);
  kpo = xid + yid*nx + kpo*nx*ny;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    U_advected = dev_conserved[(n_fields-1)*n_cells + id];
    U_total = E - 0.5*d*( vx*vx + vy*vy + vz*vz );

    //find the max nearby total energy
    Emax = fmax(dev_conserved[4*n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4*n_cells + ipo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jmo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jpo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + kmo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + kpo]);

    if (U_total/Emax > eta_2 ) U = U_total;
    else U = U_advected;

    //Optional: Avoid Negative Internal  Energies
    U = fmax(U, (Real) TINY_NUMBER);

    //Write Selected internal energy to the GasEnergy array ONLY
    //to avoid mixing updated and non-updated values of E
    //since the Dual Energy condition depends on the neighbour cells
    dev_conserved[(n_fields-1)*n_cells + id] = U;

  }
}

__global__ void Sync_Energies_1D(Real *dev_conserved, int nx, int n_ghost, Real gamma, int n_fields)
{
  int id, xid, n_cells;
  Real d, d_inv, vx, vy, vz, U;
  n_cells = nx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  xid = id;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    U = dev_conserved[(n_fields-1)*n_cells + id];

    //Use the previously selected Internal Energy to update the total energy
    dev_conserved[4*n_cells + id] = 0.5*d*( vx*vx + vy*vy + vz*vz ) + U;
  }

}


__global__ void Sync_Energies_2D(Real *dev_conserved, int nx, int ny, int n_ghost, Real gamma, int n_fields)
{
  int id, xid, yid, n_cells;
  Real d, d_inv, vx, vy, vz, U;
  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    U = dev_conserved[(n_fields-1)*n_cells + id];

    //Use the previously selected Internal Energy to update the total energy
    dev_conserved[4*n_cells + id] = 0.5*d*( vx*vx + vy*vy + vz*vz ) + U;
  }

}


__global__ void Sync_Energies_3D(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real gamma, int n_fields)
{
  //Called in a separate kernel to avoid interfering with energy selection in Select_Internal_Energy

  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, U;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    U = dev_conserved[(n_fields-1)*n_cells + id];

    //Use the previously selected Internal Energy to update the total energy
    dev_conserved[4*n_cells + id] = 0.5*d*( vx*vx + vy*vy + vz*vz ) + U;
  }
}


#endif //DE

#ifdef TEMPERATURE_FLOOR
__global__ void Apply_Temperature_Floor(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields,  Real U_floor )
{
  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, E, Ekin, U;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    Ekin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );

    U = ( E - Ekin ) / d;
    if ( U < U_floor ) dev_conserved[4*n_cells + id] = Ekin + d*U_floor;

    #ifdef DE
    U = dev_conserved[(n_fields-1)*n_cells + id] / d ;
    if ( U < U_floor ) dev_conserved[(n_fields-1)*n_cells + id] = d*U_floor ;
    #endif
  }
}
#endif //TEMPERATURE_FLOOR


__device__ Real Average_Cell_Single_Field( int field_indx, int i, int j, int k, int nx, int ny, int nz, int ncells, Real *conserved ){
  Real v_l, v_r, v_d, v_u, v_b, v_t, v_avrg;
  int id;

  id = (i-1) + (j)*nx + (k)*nx*ny;
  v_l = conserved[ field_indx*ncells + id ];
  id = (i+1) + (j)*nx + (k)*nx*ny;
  v_r = conserved[ field_indx*ncells + id ];
  id = (i) + (j-1)*nx + (k)*nx*ny;
  v_d = conserved[ field_indx*ncells + id ];
  id = (i) + (j+1)*nx + (k)*nx*ny;
  v_u = conserved[ field_indx*ncells + id ];
  id = (i) + (j)*nx + (k-1)*nx*ny;
  v_b = conserved[ field_indx*ncells + id ];
  id = (i) + (j)*nx + (k+1)*nx*ny;
  v_t = conserved[ field_indx*ncells + id ];
  v_avrg = ( v_l + v_r + v_d + v_u + v_b + v_t ) / 6;
  id = (i) + (j)*nx + (k)*nx*ny;
  conserved[ field_indx*ncells + id ] = v_avrg;
  return v_avrg;

}

__device__ void Average_Cell_All_Fields( int i, int j, int k, int nx, int ny, int nz, int ncells, int n_fields, Real *conserved ){

  // Average Density
  Average_Cell_Single_Field( 0, i, j, k, nx, ny, nz, ncells, conserved );
  // Average Momentum_x
  Average_Cell_Single_Field( 1, i, j, k, nx, ny, nz, ncells, conserved );
  // Average Momentum_y
  Average_Cell_Single_Field( 2, i, j, k, nx, ny, nz, ncells, conserved );
  // Average Momentum_z
  Average_Cell_Single_Field( 3, i, j, k, nx, ny, nz, ncells, conserved );
  // Average Energy
  Average_Cell_Single_Field( 4, i, j, k, nx, ny, nz, ncells, conserved );
  #ifdef  MHD
    // Average MHD
    Average_Cell_Single_Field( 5+NSCALARS, i,   j,   k,   nx, ny, nz, ncells, conserved );
    Average_Cell_Single_Field( 6+NSCALARS, i,   j,   k,   nx, ny, nz, ncells, conserved );
    Average_Cell_Single_Field( 7+NSCALARS, i,   j,   k,   nx, ny, nz, ncells, conserved );
    Average_Cell_Single_Field( 5+NSCALARS, i-1, j,   k,   nx, ny, nz, ncells, conserved );
    Average_Cell_Single_Field( 6+NSCALARS, i,   j-1, k,   nx, ny, nz, ncells, conserved );
    Average_Cell_Single_Field( 7+NSCALARS, i,   j,   k-1, nx, ny, nz, ncells, conserved );
  #endif  //MHD
  #ifdef DE
  // Average GasEnergy
  Average_Cell_Single_Field( n_fields-1, i, j, k, nx, ny, nz, ncells, conserved );
  #endif  //DE
}


#endif //CUDA
