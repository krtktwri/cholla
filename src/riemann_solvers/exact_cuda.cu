
#include <hip/hip_runtime.h>
/*! \file exact_cuda.cu
 *  \brief Function definitions for the cuda exact Riemann solver.*/

#ifdef CUDA

#include "../utils/gpu.hpp"
#include <math.h>
#include <stdio.h>
#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../riemann_solvers/exact_cuda.h"

#ifdef DE //PRESSURE_DE
#include "../utils/hydro_utilities.h"
#endif



/*! \fn Calculate_Exact_Fluxes_CUDA(Real *dev_bounds_L, Real *dev_bounds_R, Real *dev_flux, int nx, int ny, int nz, int n_ghost, Real gamma, int dir, int n_fields)
 *  \brief Exact Riemann solver based on the Fortran code given in Sec. 4.9 of Toro (1999). */
__global__ void Calculate_Exact_Fluxes_CUDA(Real *dev_bounds_L, Real *dev_bounds_R, Real *dev_flux, int nx, int ny, int nz, int n_ghost, Real gamma, int dir, int n_fields)
{
  // get a thread index
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;

  int n_cells = nx*ny*nz;
  int o1, o2, o3;
  if (dir == 0) {
    o1 = 1; o2 = 2; o3 = 3;
  }
  if (dir == 1) {
    o1 = 2; o2 = 3; o3 = 1;
  }
  if (dir == 2) {
    o1 = 3; o2 = 1; o3 = 2;
  }

  Real dl, vxl, vyl, vzl, pl, cl; //density, velocity, pressure, sound speed (left)
  Real dr, vxr, vyr, vzr, pr, cr; //density, velocity, pressure, sound speed (right)
  Real ds, vs, ps, Es; //sample_CUDAd density, velocity, pressure, total energy
  Real vm, pm; //velocity and pressure in the star region

  #ifdef DE
  Real gel, ger, E_kin, E, dge ;
  #endif

  #ifdef SCALAR
  Real scalarl[NSCALARS], scalarr[NSCALARS];
  #endif


  // Each thread executes the solver independently
  //if (xid > n_ghost-3 && xid < nx-n_ghost+1 && yid < ny && zid < nz)
  if (xid < nx && yid < ny && zid < nz)
  {
    // retrieve primitive variables
    dl  = dev_bounds_L[            tid];
    vxl = dev_bounds_L[o1*n_cells + tid]/dl;
    vyl = dev_bounds_L[o2*n_cells + tid]/dl;
    vzl = dev_bounds_L[o3*n_cells + tid]/dl;
    #ifdef DE //PRESSURE_DE
    E = dev_bounds_L[4*n_cells + tid];
    E_kin = 0.5 * dl * ( vxl*vxl + vyl*vyl + vzl*vzl );
    dge = dev_bounds_L[(n_fields-1)*n_cells + tid];
    pl = hydro_utilities::Get_Pressure_From_DE( E, E - E_kin, dge, gamma );
    #else
    pl  = (dev_bounds_L[4*n_cells + tid] - 0.5*dl*(vxl*vxl + vyl*vyl + vzl*vzl)) * (gamma - 1.0);
    #endif //PRESSURE_DE
    pl  = fmax(pl, (Real) TINY_NUMBER);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      scalarl[i] = dev_bounds_L[(5+i)*n_cells + tid]/dl;
    }
    #endif
    #ifdef DE
    gel = dge / dl;
    #endif
    dr  = dev_bounds_R[            tid];
    vxr = dev_bounds_R[o1*n_cells + tid]/dr;
    vyr = dev_bounds_R[o2*n_cells + tid]/dr;
    vzr = dev_bounds_R[o3*n_cells + tid]/dr;
    #ifdef DE //PRESSURE_DE
    E = dev_bounds_R[4*n_cells + tid];
    E_kin = 0.5 * dr * ( vxr*vxr + vyr*vyr + vzr*vzr );
    dge = dev_bounds_R[(n_fields-1)*n_cells + tid];
    pr = Get_Pressure_From_DE( E, E - E_kin, dge, gamma );
    #else
    pr  = (dev_bounds_R[4*n_cells + tid] - 0.5*dr*(vxr*vxr + vyr*vyr + vzr*vzr)) * (gamma - 1.0);
    #endif //PRESSURE_DE
    pr  = fmax(pr, (Real) TINY_NUMBER);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      scalarr[i] = dev_bounds_R[(5+i)*n_cells + tid]/dr;
    }
    #endif
    #ifdef DE
    ger = dge / dr;
    #endif


    // compute sounds speeds in left and right regions
    cl = sqrt(gamma * pl / dl);
    cr = sqrt(gamma * pr / dr);

    // test for the pressure positivity condition
    if ((2.0 / (gamma - 1.0)) * (cl+cr) <= (vxr-vxl))
    {
      // the initial data is such that vacuum is generated
      printf("Vacuum is generated by the initial data.\n");
      printf("%f %f %f %f %f %f\n", dl, vxl, pl, dr, vxr, pr);
    }

    // Find the exact solution for pressure and velocity in the star region
    starpv_CUDA(&pm, &vm, dl, vxl, pl, cl, dr, vxr, pr, cr, gamma);

    //sample_CUDA the solution at the cell interface
    sample_CUDA(pm, vm, &ds, &vs, &ps, dl, vxl, pl, cl, dr, vxr, pr, cr, gamma);

    // calculate the fluxes through the cell interface
    dev_flux[tid] = ds*vs;
    dev_flux[o1*n_cells + tid] = ds*vs*vs+ps;
    if (vs >= 0)
    {
      dev_flux[o2*n_cells + tid] = ds*vs*vyl;
      dev_flux[o3*n_cells + tid] = ds*vs*vzl;
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        dev_flux[(5+i)*n_cells + tid] = ds*vs*scalarl[i];
      }
      #endif
      #ifdef DE
      dev_flux[(n_fields-1)*n_cells + tid] = ds*vs*gel;
      #endif
      Es = (ps/(gamma - 1.0)) + 0.5*ds*(vs*vs + vyl*vyl + vzl*vzl);
    }
    else
    {
      dev_flux[o2*n_cells + tid] = ds*vs*vyr;
      dev_flux[o3*n_cells + tid] = ds*vs*vzr;
      #ifdef SCALAR
      for (int i=0; i<NSCALARS; i++) {
        dev_flux[(5+i)*n_cells + tid] = ds*vs*scalarr[i];
      }
      #endif
      #ifdef DE
      dev_flux[(n_fields-1)*n_cells + tid] = ds*vs*ger;
      #endif
      Es = (ps/(gamma - 1.0)) + 0.5*ds*(vs*vs + vyr*vyr + vzr*vzr);
    }
    dev_flux[4*n_cells + tid] = (Es+ps)*vs;
  }

}


__device__ Real guessp_CUDA(Real dl, Real vxl, Real pl, Real cl, Real dr, Real vxr, Real pr, Real cr, Real gamma)
{
  // purpose:  to provide a guessed value for pressure
  //    pm in the Star Region. The choice is made
  //    according to adaptive Riemann solver using
  //    the PVRS and TSRS approximate Riemann
  //    solvers. See Sect. 9.5 of Toro (1999)

  Real gl, gr, ppv, pm;
  const Real TOL = 1.0e-6;

  // compute guess pressure from PVRS Riemann solver
  ppv = 0.5*(pl + pr) + 0.125*(vxl - vxr)*(dl + dr)*(cl + cr);

  if (ppv < 0.0) ppv = 0.0;
  // Two-Shock Riemann solver with PVRS as estimate
  gl = sqrt((2.0 / ((gamma + 1.0)*dl))/(((gamma - 1.0) / (gamma + 1.0))*pl + ppv));
  gr = sqrt((2.0 / ((gamma + 1.0)*dr))/(((gamma - 1.0) / (gamma + 1.0))*pr + ppv));
  pm = (gl*pl + gr*pr - (vxr-vxl))/(gl + gr);

  if (pm < 0.0) pm = TOL;

  return pm;
}


__device__ void prefun_CUDA(Real *f, Real *fd, Real p, Real dk, Real pk, Real ck, Real gamma)
{
  // purpose:  to evaluate the pressure functions
  // fl and fr in the exact Riemann solver
  // and their first derivatives

  Real qrt;

  if (p <= pk) {
    // rarefaction wave
    *f = (2.0 / (gamma - 1.0))*ck*(powf(p/pk, (gamma - 1.0)/(2.0 * gamma)) - 1.0);
    *fd = (1.0/(dk*ck))*powf((p/pk), -((gamma + 1.0)/(2.0 * gamma)));
  }
  else
  {
    // shock wave
    qrt = sqrt(((2.0 / (gamma + 1.0)) / dk)/((((gamma - 1.0) / (gamma + 1.0)) * pk) + p));
    *f = (p - pk)*qrt;
    *fd = (1.0 - 0.5*(p - pk)/((((gamma - 1.0) / (gamma + 1.0)) * pk) + p))*qrt;
  }
}


__device__ void starpv_CUDA(Real *p, Real *v, Real dl, Real vxl, Real pl, Real cl, Real dr, Real vxr, Real pr, Real cr, Real gamma)
{
  // purpose:  Uses Newton-Raphson iteration
  // to compute the solution for pressure and
  // velocity in the Star Region

  const int nriter = 20;
  const Real TOL = 1.0e-6;
  Real change, fl, fld, fr, frd, pold, pstart;

  //guessed value pstart is computed
  pstart = guessp_CUDA(dl, vxl, pl, cl, dr, vxr, pr, cr, gamma);
  pold = pstart;

  int i = 0;
  for (i=0 ; i <= nriter; i++) {
    prefun_CUDA(&fl, &fld, pold, dl, pl, cl, gamma);
    prefun_CUDA(&fr, &frd, pold, dr, pr, cr, gamma);
    *p = pold - (fl + fr + vxr - vxl)/(fld + frd);
    change = 2.0*fabs((*p - pold)/(*p + pold));

    if (change <= TOL) break;
    if (*p < 0.0) *p = TOL;
    pold = *p;
  }
  if (i > nriter) {
    //printf("Divergence in Newton-Raphson iteration. p = %e\n", *p);
  }

  // compute velocity in star region
  *v = 0.5*(vxl + vxr + fr - fl);

}


__device__ void sample_CUDA(const Real pm, const Real vm, Real *d, Real *v, Real *p,
      Real dl, Real vxl, Real pl, Real cl, Real dr, Real vxr, Real pr, Real cr, Real gamma)
{
  // purpose:  to sample the solution throughout the wave
  //   pattern. Pressure pm and velocity vm in the
  //   star region are known. Sampled
  //   values are d, v, p.

  Real c, sl, sr;

  if (vm >= 0) // sampling point lies to the left of the contact discontinuity
  {
    if (pm <= pl) // left rarefaction
    {
      if (vxl - cl >= 0) // sampled point is in left data state
      {
        *d = dl;
        *v = vxl;
        *p = pl;
      }
      else
      {
        if (vm - cl*powf(pm/pl, (gamma - 1.0)/(2.0 * gamma)) < 0) // sampled point is in star left state
        {
          *d = dl*powf(pm/pl, 1.0/gamma);
          *v = vm;
          *p = pm;
        }
        else // sampled point is inside left fan
        {
          c = (2.0 / (gamma + 1.0))*(cl + ((gamma - 1.0) / 2.0)*vxl);
          *v = c;
          *d = dl*powf(c/cl, 2.0 / (gamma - 1.0));
          *p = pl*powf(c/cl, 2.0 * gamma / (gamma - 1.0));
        }
      }
    }
    else // left shock
    {
      sl = vxl - cl*sqrt(((gamma + 1.0)/(2.0 * gamma))*(pm/pl) + ((gamma - 1.0)/(2.0 * gamma)));
      if (sl >= 0) // sampled point is in left data state
      {
        *d = dl;
        *v = vxl;
        *p = pl;
      }
      else // sampled point is in star left state
      {
        *d = dl*(pm/pl + ((gamma - 1.0) / (gamma + 1.0)))/((pm/pl)*((gamma - 1.0) / (gamma + 1.0)) + 1.0);
        *v = vm;
        *p = pm;
      }
    }
  }
  else // sampling point lies to the right of the contact discontinuity
  {
    if (pm > pr) // right shock
    {
      sr = vxr + cr*sqrt(((gamma + 1.0)/(2.0 * gamma))*(pm/pr) + ((gamma - 1.0)/(2.0 * gamma)));
      if (sr <= 0) // sampled point is in right data state
      {
        *d = dr;
        *v = vxr;
        *p = pr;
      }
      else // sampled point is in star right state
      {
        *d = dr*(pm/pr + ((gamma - 1.0) / (gamma + 1.0)))/((pm/pr)*((gamma - 1.0) / (gamma + 1.0)) + 1.0);
        *v = vm;
        *p = pm;
      }
    }
    else // right rarefaction
    {
      if (vxr + cr <= 0) // sampled point is in right data state
      {
        *d = dr;
        *v = vxr;
        *p = pr;
      }
      else
      {
        if (vm + cr*powf(pm/pr, (gamma - 1.0)/(2.0 * gamma)) >= 0) // sampled point is in star right state
        {
          *d = dr*powf(pm/pr, (1.0/gamma));
          *v = vm;
          *p = pm;
        }
        else // sampled point is inside right fan
        {
          c = (2.0 / (gamma + 1.0))*(cr - ((gamma - 1.0) / 2.0)*vxr);
          *v = -c;
          *d = dr*powf(c/cr, 2.0 / (gamma - 1.0));
          *p = pr*powf(c/cr, 2.0 * gamma / (gamma - 1.0));
        }
      }
    }
  }
}



#endif //CUDA
